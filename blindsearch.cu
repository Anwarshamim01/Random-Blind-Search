#include <hip/hip_runtime.h>

#include <cmath>
#include <cstdio>
#include <hiprand/hiprand.h>         // For host-side cuRAND functions
#include <hiprand/hiprand_kernel.h>  // For device-side cuRAND functions and state types
#include <float.h>          // For FLT_EPSILON or DBL_EPSILON

// Helper function for checking CUDA Runtime API errors
#define CHECK_CUDA_ERROR(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
struct RunData {
    int run_index;
    int first_return_time;
    int first_hit_time;
    // Note: `visited_targets` is not included here due to complexity.
};
// Helper function for checking cuRAND API errors
#define CHECK_CURAND_ERROR(ans) { curandAssert((ans), __FILE__, __LINE__); }
inline void curandAssert(hiprandStatus_t code, const char *file, int line, bool abort=true)
{
   if (code != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr,"cuRANDAssert: %d (Error code) in %s %d\n", code, file, line); // cuRAND doesn't have string messages like CUDA
      if (abort) exit(code);
   }
}


// Periodic boundary function for device
__device__ inline float periodic_boundaries_d(float pos, float boundary) {
    // Equivalent to Python's ((pos + boundary) % (2 * boundary)) - boundary
    // Handles potential negative results from fmodf
    float result = fmodf(pos + boundary, 2.0f * boundary);
    if (result < 0) {
        result += 2.0f * boundary;
    }
    return result - boundary;
}

// Alpha-stable Levy step generation on device (Zolatorev/Mantegna-type method)
// Using double precision for intermediate calculations for better accuracy
// Takes the specific Philox state type
__device__ inline double levy_alpha_stable_d(hiprandStatePhilox4_32_10_t* rng_state, double alpha, double beta, double mu, double c, double delta_t) {
    // theta ~ U(-pi/2, pi/2)
    double theta = hiprand_uniform_double(rng_state) * M_PI - M_PI_2;

    // W ~ Exp(1) generated from Uniform using inverse transform: W = -log(U)
    double U_exp = hiprand_uniform_double(rng_state);
    // Ensure U_exp > 0 to avoid log(0)
    U_exp = fmax(U_exp, DBL_EPSILON); // Using fmax with double precision
    double W = -log(U_exp);

    // Ensure W is not zero to avoid division by zero
    W = fmax(W, 1e-10); // Using fmax with double precision


    double num = sin(alpha * theta);
    double den = pow(cos(theta), 1.0 / alpha);
    double cos_term = cos((1.0 - alpha) * theta);

    double correction_factor;
    // Handle potential negative base for pow, though for alpha in (0, 2), this shouldn't be an issue
    // for cos_term >= 0. If cos_term < 0, the Levy formula derivation typically implies alpha=1 or specific cases.
    // Assuming the Python logic handles this correctly by using abs implicitly or the formula structure prevents it.
    // Let's follow the Python's likely intent for this specific formula structure.
    // Ensure the base of pow is non-negative
    double pow_base = fabs(cos_term) / W;
    if (pow_base < 0) pow_base = 0; // Should not happen with fabs, but safeguard

    double exponent = (1.0 - alpha) / alpha;

    correction_factor = pow(pow_base, exponent);


    double step_length = (num / den) * correction_factor;

    if (isnan(step_length) || isinf(step_length)) {
        step_length = 0.0;
    }

    return c * step_length + mu * delta_t;
}

// CUDA Kernel for simulating Levy flights
__global__ void simulate_levy_runs_kernel(
    int num_runs,
    int max_time,
    float alpha, float beta, float mu, float c, float delta_t,
    float vision_radius_sq, // Squared vision radius for faster comparison
    float boundary,
    const float* d_targets, // Device pointer to target positions (x, y pairs)
    int num_targets,
    int starting_target_index, // Index of the starting target
    const float* d_starting_pos, // Device pointer to starting position (x, y)
    int* d_first_return_time,   // Device pointer for output (initialized to -1)
    int* d_first_hit_time,      // Device pointer for output (initialized to -1)
    hiprandStatePhilox4_32_10_t* rng_states  // Device pointer for RNG states (using specific type)
) {
    int run_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (run_idx >= num_runs) {
        return;
    }

    // Load random number generator state for this thread
    hiprandStatePhilox4_32_10_t rng_state = rng_states[run_idx];

    float walker_pos_x = d_starting_pos[0];
    float walker_pos_y = d_starting_pos[1];

    int step_count = 0;
    int first_return_time = -1;
    int first_hit_time = -1;

    // Simulation loop
    while (step_count < max_time) { // Continue as long as max_time not reached

        // Generate Levy step using double precision internally
        double jump_length_double = fabs(levy_alpha_stable_d(&rng_state, alpha, beta, mu, c, delta_t));
        double angle = hiprand_uniform_double(&rng_state) * 2.0 * M_PI;

        double dx_total_double = jump_length_double * cos(angle);
        double dy_total_double = jump_length_double * sin(angle);

        // Determine number of small increments
        // The Python code has a loop to ensure increments are <= 1.0
        // Lets replicate that logic, ensuring at least one increment
        double dd_double = sqrt(dx_total_double*dx_total_double + dy_total_double*dy_total_double);

        // Replace fmax(1, (int)dd_double) with a manual check
        int num_increments = (int)dd_double;
        if (num_increments < 1) {
            num_increments = 1;
        }

        // Adjust num_increments until step size per increment is <= 1.0
        // Avoid infinite loops if dd_double is huge (e.g., from an extreme Levy step)
        if (dd_double > 1e15) { // Add a safeguard for extremely large jumps
            num_increments = (int)dd_double; // Just use dd as num_increments in extreme cases
            if (num_increments <= 0) num_increments = 1; // Ensure at least 1
        } else {
             // Only perform the while loop adjustment if not an extremely large jump
             // The initial check `if (num_increments < 1) num_increments = 1;` handles dd_double <= 1.0 cases
             while (num_increments > 0 && dd_double / num_increments > 1.0) {
                 num_increments++;
            }
            // Re-check after while loop just in case num_increments somehow became 0
            if (num_increments == 0) num_increments = 1;
        }

        // Handle case where dd_double is 0 (no jump)
        if (dd_double == 0) num_increments = 1;


        float dx_step = dx_total_double / num_increments;
        float dy_step = dy_total_double / num_increments;

        // Loop over increments
        for (int i = 0; i < num_increments; ++i) {
            if (step_count >= max_time) {
                break; // Break inner loop if max_time is reached during increments
            }

            walker_pos_x += dx_step;
            walker_pos_y += dy_step;

            // Apply periodic boundaries
            walker_pos_x = periodic_boundaries_d(walker_pos_x, boundary);
            walker_pos_y = periodic_boundaries_d(walker_pos_y, boundary);

            step_count++;

            // Check for target detection (linear scan for nearest target)
            // This is the most performance-critical part and could be optimized
            float min_dist_sq = 1e18; // Initialize with a large value
            int nearest_target_idx = -1;

            for (int j = 0; j < num_targets; ++j) {
                float target_x = d_targets[j * 2];
                float target_y = d_targets[j * 2 + 1];
                float dist_sq = (walker_pos_x - target_x) * (walker_pos_x - target_x) +
                                (walker_pos_y - target_y) * (walker_pos_y - target_y);

                if (dist_sq < min_dist_sq) {
                    min_dist_sq = dist_sq;
                    nearest_target_idx = j;
                }
            }

            // Check if detected and record first times
            if (nearest_target_idx != -1 && min_dist_sq <= vision_radius_sq) {
                if (nearest_target_idx == starting_target_index && step_count > 60 && first_return_time == -1) {
                    first_return_time = step_count;
                } else if (nearest_target_idx != starting_target_index && first_hit_time == -1) {
                    first_hit_time = step_count;
                }
            }

             // Stop simulation for this run if both events have occurred
            if (first_return_time != -1 && first_hit_time != -1) {
                break; // Break inner loop
            }
        } // End of inner loop (increments)

        // Break outer loop if both events have occurred or max_time is reached
        if (first_return_time != -1 && first_hit_time != -1) {
            break; // Break outer loop
        }
         if (step_count >= max_time) {
            break; // Break outer loop if max_time reached
        }
    } // End of simulation loop

    // Store results in global memory
    d_first_return_time[run_idx] = first_return_time;
    d_first_hit_time[run_idx] = first_hit_time;

    // Save the updated RNG state (optional for independent runs)
    // rng_states[run_idx] = rng_state; // Not strictly needed if state is init per kernel launch
}

// Kernel to initialize device RNG states
__global__ void init_rng_states_kernel(hiprandStatePhilox4_32_10_t* states, unsigned long long base_seed, int num_threads) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_threads) {
        // Use thread index as sequence, base_seed as initial seed
        hiprand_init(base_seed, idx, 0, &states[idx]);
    }
}


int main() {
    // Parameters (matching Python code)
    float alpha = 0.6f;
    float beta = 0.0f;
    float mu = 0.0f;
    float c = 1.0f;
    float delta_t = 1.0f; // Not directly used in step calculation magnitude in Python, but kept for structure
    int max_time = 100000;
    float vision_radius = 3.0f;
    float vision_radius_sq = vision_radius * vision_radius; // Use squared radius
    int num_runs =200000;
    float boundary = 12000.0f;
    int num_targets = 1200;

    // --- Target setup ---
    int grid_size = static_cast<int>(sqrt(num_targets));
    // Ensure num_targets is a perfect square or handle grid size appropriately
    if (grid_size * grid_size != num_targets) {
         fprintf(stderr, "Warning: num_targets is not a perfect square. Adjusting grid_size.\n");
         // Simple approximation, might not perfectly match Python's grid
         grid_size = static_cast<int>(sqrt(num_targets));
         // To get exactly num_targets points, a different strategy might be needed
         // For this conversion, assume grid_size * grid_size is close to num_targets
         // and generate grid_size*grid_size points.
         num_targets = grid_size * grid_size;
         fprintf(stderr, "Adjusted num_targets to %d based on grid_size %d.\n", num_targets, grid_size);
    }

    float* targets = (float*)malloc(num_targets * 2 * sizeof(float)); // x, y pairs
    if (targets == NULL) {
        fprintf(stderr, "Failed to allocate host memory for targets.\n");
        return 1;
    }

    float x_coords[grid_size];
    float y_coords[grid_size];

    for (int i = 0; i < grid_size; ++i) {
        x_coords[i] = -boundary + (2.0f * boundary / (grid_size - 1)) * i;
        y_coords[i] = -boundary + (2.0f * boundary / (grid_size - 1)) * i;
    }

    int target_idx = 0;
    for (int i = 0; i < grid_size; ++i) {
        for (int j = 0; j < grid_size; ++j) {
            targets[target_idx * 2] = x_coords[i];
            targets[target_idx * 2 + 1] = y_coords[j];
            target_idx++;
        }
    }

    // --- Starting position ---
    int middle_index = (grid_size / 2) * grid_size + (grid_size / 2);
    float starting_pos[2];
    starting_pos[0] = targets[middle_index * 2];
    starting_pos[1] = targets[middle_index * 2 + 1];

    // --- CUDA Setup ---
    float* d_targets;
    float* d_starting_pos;
    int* d_first_return_time;
    int* d_first_hit_time;
    hiprandStatePhilox4_32_10_t* rng_states; // Use specific state type

    // Allocate device memory
    CHECK_CUDA_ERROR(hipMalloc(&d_targets, num_targets * 2 * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_starting_pos, 2 * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc(&d_first_return_time, num_runs * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc(&d_first_hit_time, num_runs * sizeof(int)));
    // Allocate memory for specific cuRAND states
    CHECK_CUDA_ERROR(hipMalloc(&rng_states, num_runs * sizeof(hiprandStatePhilox4_32_10_t)));

    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpy(d_targets, targets, num_targets * 2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_starting_pos, starting_pos, 2 * sizeof(float), hipMemcpyHostToDevice));

    // Initialize output arrays on device to -1 (or another sentinel value)
    CHECK_CUDA_ERROR(hipMemset(d_first_return_time, -1, num_runs * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemset(d_first_hit_time, -1, num_runs * sizeof(int)));

    // --- Initialize Device RNG States Kernel ---
    // This kernel initializes each thread's RNG state
    // using its thread index as the sequence number.
    unsigned long long seed = 12345ULL; // Base seed for reproducibility

    int threads_per_block_init = 512;
    int num_blocks_init = (num_runs + threads_per_block_init - 1) / threads_per_block_init;

    printf("Initializing device RNG states...\n");
    init_rng_states_kernel<<<num_blocks_init, threads_per_block_init>>>(rng_states, seed, num_runs);
    CHECK_CUDA_ERROR(hipGetLastError()); // Check for errors from the init kernel launch
    CHECK_CUDA_ERROR(hipDeviceSynchronize()); // Wait for init kernel

    printf("RNG states initialized.\n");

    // --- Simulation Kernel Launch ---
    int threads_per_block = 256;
    int num_blocks = (num_runs + threads_per_block - 1) / threads_per_block;

    printf("Launching simulation kernel with %d blocks and %d threads per block for %d runs.\n", num_blocks, threads_per_block, num_runs);

    simulate_levy_runs_kernel<<<num_blocks, threads_per_block>>>(
        num_runs,
        max_time,
        alpha, beta, mu, c, delta_t,
        vision_radius_sq,
        boundary,
        d_targets,
        num_targets,
        middle_index, // Pass starting target index
        d_starting_pos,
        d_first_return_time,
        d_first_hit_time,
        rng_states // Pass the device array of initialized states
    );

    // Check for kernel launch errors
    CHECK_CUDA_ERROR(hipGetLastError());
    // Wait for the kernel to complete
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    printf("Simulation kernel finished.\n");

    // --- Retrieve Results ---
    int* h_first_return_time = (int*)malloc(num_runs * sizeof(int));
    int* h_first_hit_time = (int*)malloc(num_runs * sizeof(int));

    if (h_first_return_time == NULL || h_first_hit_time == NULL) {
        fprintf(stderr, "Failed to allocate host memory for results.\n");
        // Clean up device memory before exiting
        hipFree(d_targets);
        hipFree(d_starting_pos);
        hipFree(d_first_return_time);
        hipFree(d_first_hit_time);
        hipFree(rng_states);
        free(targets);
        free(h_first_return_time); // Free if one failed but the other succeeded
        free(h_first_hit_time);
        return 1;
    }


    CHECK_CUDA_ERROR(hipMemcpy(h_first_return_time, d_first_return_time, num_runs * sizeof(int), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipMemcpy(h_first_hit_time, d_first_hit_time, num_runs * sizeof(int), hipMemcpyDeviceToHost));

    // --- Process Results (optional, matching Python's print) ---
    printf("\nSimulation Results Summary (first 10 and last 10 runs):\n");
    for (int i = 0; i < num_runs; ++i) {
        // Only print a few to avoid excessive output, or print a summary
        if (i < 10 || i >= num_runs - 10) {
             printf("Run %d: First Return = %d, First Hit = %d\n",
                i, h_first_return_time[i], h_first_hit_time[i]);
        }
    }

    // You could save h_first_return_time and h_first_hit_time to a file here,
    // similar to the Python pickle.
    // --- Save Results to Binary File ---
    const char* filename = "simulation_results.pickle";
    FILE* fp = fopen(filename, "wb");
    if (!fp) {
        fprintf(stderr, "Failed to open file %s for writing.\n", filename);
    } else {
        // Write the total number of runs first
        fwrite(&num_runs, sizeof(int), 1, fp);

        // Then write each run's data
        for (int i = 0; i < num_runs; ++i) {
            RunData rd;
            rd.run_index = i;
            rd.first_return_time = h_first_return_time[i];
            rd.first_hit_time = h_first_hit_time[i];

            fwrite(&rd, sizeof(RunData), 1, fp);
        }

        fclose(fp);
        printf("Results saved to %s\n", filename);
    }
    // --- Cleanup ---
    hipFree(d_targets);
    hipFree(d_starting_pos);
    hipFree(d_first_return_time);
    hipFree(d_first_hit_time);
    hipFree(rng_states); // Free cuRAND states device memory

    // No host generator handle is created/destroyed in this version with init_rng_states_kernel

    free(targets);
    free(h_first_return_time);
    free(h_first_hit_time);

    printf("\nCleanup complete. Simulation finished.\n");

    return 0;
}
